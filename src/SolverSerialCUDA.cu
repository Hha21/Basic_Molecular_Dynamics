#include "hip/hip_runtime.h"
#include "SolverSerialCUDA.h"

///Boltzmann Constant k_b
static constexpr double BOLTZMANN = 0.8314459920816467;

// GPU STRUCT FOR PROPERTIES (CANNOT ACCESS STATIC MEMORY)
struct GPU_ParticleProp {
    double mass[2];
    double epsilon[2][2];
    double sigma6[2][2];
    double sigma12[2][2];
};

//DECLARE GPU CONST
__constant__ GPU_ParticleProp GPU_PARTICLE_PROPERTIES;

std::array<double, 3> Solver::getRandPos() {
    //GET {Lx, Ly, Lz} ONCE
    static const std::array<double, 3> dims = this->domain.getDims();

    return  {dims[0] * (Solver::randDouble()),
            dims[1] * (Solver::randDouble()),
            dims[2] * (Solver::randDouble())};
}

std::array<double, 3> Solver::getRandVel() {
    return {-0.5 + (Solver::randDouble()),
            -0.5 + (Solver::randDouble()),
            -0.5 + (Solver::randDouble())};
}

bool Solver::isValidPos(const std::array<double, 3>& pos) {
    // IF NO PARTICLES THEN ADD
    if (this->particles.empty()) return true;

    static const double minDist2 = 0.5 * 0.5;

    for (const Particle& particle : this->particles) {
        const std::array<double, 3> pPos = particle.getPos();
        double dist2 =  (pos[0] - pPos[0]) * (pos[0] - pPos[0]) +
                    (pos[1] - pPos[1]) * (pos[1] - pPos[1]) +
                    (pos[2] - pPos[2]) * (pos[2] - pPos[2]);
        if (dist2 < minDist2) return false;
    }
    return true;
}

/**
 * @brief Constructs the solver, intiliases problem and runs upon being called.
 */
Solver::Solver(double Lx_, double Ly_, double Lz_, 
                double dt_, double T_, double temp_, 
                double percType1_, unsigned int N_, ICScenario scenario_) 
        :   domain(Lx_, Ly_, Lz_),
            dt(dt_), T(T_), temp(temp_),
            percType1(percType1_), N(N_),
            time(0.0), scenario(scenario_),
            randGen(std::random_device{}()), 
            distribution(0.0, 1.0),
            logger("kinetic_energy.txt",                                    // IF RAND -> NO particles.txt
                scenario == ICScenario::RANDOM ? "" : "particles.txt") {

            Solver::initParticles();
            Solver::allocGPUMemory();
            Solver::initPointers();
            
            std::cout << "CUDA SOLVER INITIALISED" << std::endl;
            Solver::run();
}

void Solver::allocGPUMemory() {

    // TEMP TO COPY PARTICLE PROPERTY VALUES
    GPU_ParticleProp TEMP_PARTICLE_PROPERTIES;

    for (int i = 0; i < 2; i++) {
        TEMP_PARTICLE_PROPERTIES.mass[i] = ParticleProp::mass[i];
        for (int j = 0; j < 2; j++) {
            TEMP_PARTICLE_PROPERTIES.epsilon[i][j] = ParticleProp::epsilon[i][j];
            TEMP_PARTICLE_PROPERTIES.sigma6[i][j] = ParticleProp::sigma6[i][j];
            TEMP_PARTICLE_PROPERTIES.sigma12[i][j] = ParticleProp::sigma12[i][j];
        }
    }

    // COPY PROPERTIES
    hipMemcpyToSymbol(HIP_SYMBOL(GPU_PARTICLE_PROPERTIES), &TEMP_PARTICLE_PROPERTIES, sizeof(GPU_ParticleProp));

    // SHARE THESE BETWEEN CPU AND GPU
    hipMallocManaged(&posX, this->N * sizeof(double));
    hipMallocManaged(&posY, this->N * sizeof(double));
    hipMallocManaged(&posZ, this->N * sizeof(double));
    hipMallocManaged(&FORCE_BUFFER, this->N * 3 * sizeof(double));
    hipMallocManaged(&types, this->N * sizeof(unsigned int));
}

__device__
void getGPUProperties(const int typ1, const int typ2, double* PROPERTIES) {

    // NOT USED 

    int typSum = typ1 + typ2;
    switch(typSum) {
        case(0): {
            PROPERTIES[0] = 3.0;
            PROPERTIES[1] = 1.0;
            PROPERTIES[2] = 1.0; 
            break;
        }
        case(1): {
            PROPERTIES[0] = 15.0;
            PROPERTIES[1] = 64.0;
            PROPERTIES[2] = 4096.0; 
            break;
        }
        case(2): {
            PROPERTIES[0] = 60.0;
            PROPERTIES[1] = 729.0;
            PROPERTIES[2] = 531441.0; 
            break;
        }
    }
}

__global__ 
void LJKernel(double* posX, double* posY, double* posZ, unsigned int* types,
                            double* FORCE_BUFFER, unsigned int N) {
    
    // COMPUTE FORCES ON i FROM ALL OTHER PARTICLES j

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)  return;

    double fx = 0.0;
    double fy = 0.0;
    double fz = 0.0;
    //double* PROPERTIES = new double[3];

    for (int j = 0; j < N; j++) {

        if (j == i) continue;

        unsigned int typ1 = types[i];
        unsigned int typ2 = types[j];

        double rx = posX[j] - posX[i];
        double ry = posY[j] - posY[i];
        double rz = posZ[j] - posZ[i];
        double r2 = rx * rx + ry * ry + rz * rz;

        double inv_r2 = 1.0 / r2;
        double inv_r4 = inv_r2 * inv_r2;
        double inv_r8 = inv_r4 * inv_r4;
        double inv_r14 = inv_r8 * inv_r4 * inv_r2;

        // getGPUProperties(typ1, typ2, PROPERTIES);

        // double eps_ij = PROPERTIES[0];
        // double sigma6_ij = PROPERTIES[1];
        // double sigma12_ij = PROPERTIES[2];

        double eps_ij = GPU_PARTICLE_PROPERTIES.epsilon[typ1][typ2];
        double sigma6_ij = GPU_PARTICLE_PROPERTIES.sigma6[typ1][typ2];
        double sigma12_ij = GPU_PARTICLE_PROPERTIES.sigma12[typ1][typ2];

        double force_mag = -24.0 * eps_ij * ((2.0 * sigma12_ij * inv_r14) - (sigma6_ij * inv_r8));

        fx += force_mag * rx;
        fy += force_mag * ry;
        fz += force_mag * rz;

        //printf("Value rx, fx, i, j = %f %f %d %d\n", rx, fx, i, j);
    }

    //delete[] PROPERTIES;

    // NOW CPU CAN ACCESS FORCE_BUFFER FOR PARTICLE i UPDATE
    FORCE_BUFFER[i * 3 + 0] = fx;
    FORCE_BUFFER[i * 3 + 1] = fy;
    FORCE_BUFFER[i * 3 + 2] = fz;
}

void Solver::computeForcesCUDA() {
    int THREADS_PER_BLOCK = 256;
    int NUM_INTERACTIONS = (this->N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int NUM_BLOCKS = (NUM_INTERACTIONS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    LJKernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(this->posX, this->posY, this->posZ, this->types, this->FORCE_BUFFER, this->N);

    hipDeviceSynchronize();
}

void Solver::cpHostToDevice() {

    //DEVICE NEEDS THE NEW POSITIONS FROM EACH PARTICLE

    for (const Particle& p: this->particles) {
        const std::array<double, 3>& pos = p.getPos();
        const unsigned int ID_ = p.getID();
        this->posX[ID_] = pos[0];
        this->posY[ID_] = pos[1];
        this->posZ[ID_] = pos[2];
    }
}

void Solver::cpDeviceToHost() {

    // HOST NEEDS THE NEW FORCES COMPUTED BY DEVICE

    for (unsigned int i = 0; i < this->N; ++i) {
        Particle& p = this->particles[i];
        p.addForceComp(0, this->FORCE_BUFFER[i * 3 + 0]);
        p.addForceComp(1, this->FORCE_BUFFER[i * 3 + 1]);
        p.addForceComp(2, this->FORCE_BUFFER[i * 3 + 2]);
    }
}

void Solver::freeGPUMem() {

    // FREE MEMORY ALLOCATED TO CPU

    hipDeviceSynchronize();

    hipFree(posX);
    hipFree(posY);
    hipFree(posZ);
    hipFree(FORCE_BUFFER);
    hipFree(types);

}


Solver::~Solver(){
    //Solver::freeGPUMem();
}

/**
 * @brief Initializes particles according to the specified initial condition scenario.
 */
void Solver::initParticles() {
    // IF PARTICLES
    particles.clear();

    //INIT PARTICLES DEPENDING UPON TEST CASE
    switch(this->scenario) {    
        case ICScenario::ONE: {

            std::array<double, 3> p0Pos = {10.0, 10.0, 10.0};
            std::array<double, 3> p0Vel = {0.0, 0.0, 0.0};
            
            particles.emplace_back(0,0,p0Pos,p0Vel);

            break;
        }

        case ICScenario::ONE_VEL: {

            std::array<double, 3> p0Pos = {10.0, 10.0, 10.0};
            std::array<double, 3> p0Vel = {5.0, 2.0, 1.0};

            particles.emplace_back(0,0,p0Pos,p0Vel);

            break;
        }

        case ICScenario::TWO: {

            std::array<double, 3> p0Pos = {8.5, 10.0, 10.0};
            std::array<double, 3> p0Vel = {0.0, 0.0, 0.0};

            std::array<double, 3> p1Pos = {11.5, 10.0, 10.0};
            // VEL SAME AS p0;

            particles.emplace_back(0, 0, p0Pos, p0Vel);
            particles.emplace_back(1, 0, p1Pos, p0Vel);

            break;
        }

        case ICScenario::TWO_PASS1: {

            std::array<double, 3> p0Pos = {8.5, 11.5, 10.0};
            std::array<double, 3> p0Vel = {0.5, 0.0, 0.0};

            std::array<double, 3> p1Pos = {11.5, 8.5, 10.0};
            std::array<double, 3> p1Vel = {-0.5, 0.0, 0.0};

            particles.emplace_back(0, 0, p0Pos, p0Vel);
            particles.emplace_back(1, 0, p1Pos, p1Vel);

            break;
        }

        case ICScenario::TWO_PASS2: {

            std::array<double, 3> p0Pos = {8.5, 11.3, 10.0};
            std::array<double, 3> p0Vel = {0.5, 0.0, 0.0};

            std::array<double, 3> p1Pos = {11.5, 8.7, 10.0};    
            std::array<double, 3> p1Vel = {-0.5, 0.0, 0.0};

            particles.emplace_back(0, 0, p0Pos, p0Vel);
            particles.emplace_back(1, 0, p1Pos, p1Vel);

            break;
        }

        case ICScenario::TWO_PASS3: {

            std::array<double, 3> p0Pos = {8.5, 11.3, 10.0};
            std::array<double, 3> p0Vel = {0.5, 0.0, 0.0};

            std::array<double, 3> p1Pos = {11.5, 8.7, 10.0};
            std::array<double, 3> p1Vel = {-0.5, 0.0, 0.0};

            particles.emplace_back(0, 1, p0Pos, p0Vel);
            particles.emplace_back(1, 1, p1Pos, p1Vel);

            break;
        }

        case ICScenario::RANDOM: {

            unsigned int type1Num = static_cast<unsigned int>(std::ceil((percType1 / 100.0) * this->N));
            unsigned int currType1 = 0;
            unsigned int ID_ = 0;

            while (this->particles.size() < this->N) {
                //GENERATE RANDOM POS THEN CHECK
                std::array<double, 3> newPos = Solver::getRandPos();
                if (Solver::isValidPos(newPos)) {
                    std::array<double, 3> newVel = Solver::getRandVel();
                    unsigned int type = 0;

                    if (currType1 < type1Num) {
                        type = 1;
                        currType1++;
                    }
                    
                    particles.emplace_back(ID_++, type, newPos, newVel);
                }
            }
            Solver::setTemp();
            break;
        }

        default:
            std::cerr << "ERR: INVALID SCENARIO!" << std::endl;
            exit(1);
    }
}


void Solver::resetParticles() {
    for (Particle& p : this->particles) {
        p.resetForce();
    }
}

void Solver::initPointers() {

    // ALLOCATE INITIAL VALUES FOR POS & VALUES FOR TYPES
    for (const Particle& p: this->particles) {
        const std::array<double, 3>& pos = p.getPos();
        const unsigned int ID_ = p.getID();

        this->posX[ID_] = pos[0];
        this->posY[ID_] = pos[1];
        this->posZ[ID_] = pos[2];
        this->types[ID_] = p.getType();
    }
}

//DYNAMICS 

void Solver::setTemp() {
    if (this->temp <= 0.0) return;

    std::cout << "TEMP APPLIED!" << std::endl;

    //GET CURRENT KE
    Solver::computeKE();

    double temp0 = (2.0 * this->KE) / (3.0 * BOLTZMANN * this->N);

    double lambda = std::sqrt(this->temp / temp0);

    for (Particle& p : this->particles) {
        std::array<double, 3> v0 = p.getVel();
        for (int k = 0; k < 3; ++k) v0[k] *= lambda;
        p.setVel(v0);
    }
}


void Solver::computeKE() {
    //INIT KE
    double KE_ = 0.0;
    for (const Particle& p : this->particles) {
        double speed = 0.0;

        for (int k = 0; k < 3; ++k) {
            speed += p.getVel()[k] * p.getVel()[k];
        }

        KE_ += 0.5 * p.getMass() * speed; 
    } 
    this->KE = KE_;  
}

void Solver::step() {
    // 1 - APPLY BC
    this->domain.applyBC(this->particles);

    // 2 - COMPUTE FORCES (RESET FORCES INSIDE)
    Solver::computeForcesCUDA();
    Solver::cpDeviceToHost();
    // 3 - EULER METHOD FOR UPDATE
    for (Particle& p : particles) {
        p.updateVel(this->dt);
        p.updatePos(this->dt);
        p.resetForce();
    }
    Solver::cpHostToDevice();

    // 4 - UPDATE t
    this->time += this->dt;
}

void Solver::run() {
    std::cout << "RUNNING SIMULATION... " << std::endl;

    double lastOutputTime = -0.1;

    while (this->time < this->T) { 
        if (time >= lastOutputTime + 0.1) {
            Solver::computeKE();
            logger.logParticleData(this->time, this->particles);
            logger.logKineticEnergy(this->time, this->KE);
            lastOutputTime = this->time;
        }

        Solver::step();
    }


    Solver::computeKE();
    logger.logParticleData(this->time, this->particles);
    logger.logKineticEnergy(this->time, this->KE);
    lastOutputTime = this->time;
    hipDeviceSynchronize();
}

const std::array<double, 3>& Solver::getFinalPosK(unsigned int ID) {
    //const std::array<double, 3> pos = this->particles[ID].getPos();
    return this->particles[ID].getPos();
}

const double Solver::getKE(){
    return this->KE;
}